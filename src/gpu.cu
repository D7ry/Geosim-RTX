#include "hip/hip_runtime.h"
#include "gpu.h"
#include <hip/hip_runtime_api.h>
#include <glm/glm.hpp>
#include <iostream>
#include <stdio.h>

#include "Camera.h"
#include "Image.h"
#include "Primitive.h"
#include "Settings.h"
#include "util/CUDAMath.h"

// #include "gpu_sdf.h"
#include "util/Math.h"

#include "util/Ray.h"

#include "render_settings.h"

__host__ void print_cuda_error() {
    auto err = hipGetLastError();
    if (err) {
        auto err_str = hipGetErrorString(err);
        printf("Last CUDA Error: %s\n", err_str);
    }
}

namespace CudaPlayground
{
__global__ void cudaHello() {
    printf("Hello World from CUDA thread [%d,%d]\n", threadIdx.x, blockIdx.x);
    glm::vec3 a(1.0f, 2.0f, 3.0f);
    glm::vec3 b(4.0f, threadIdx.x, blockIdx.x);
    glm::vec3 c = a + b;
    printf("c = [%f, %f, %f]\n", c.x, c.y, c.z);
}

void play() {
    std::cout << "Running CUDA Playground" << std::endl;
    int numBlocks = 16;
    int threadsPerBlock = 16;
    cudaHello<<<numBlocks, threadsPerBlock>>>();
    hipDeviceSynchronize();

    std::cout << "CUDA Playground finished" << std::endl;
}
} // namespace CudaPlayground

namespace CUDAStruct
{

__host__ CubeMap* loadCubeMap(const char* filename) {
    int channels;
    int width, height;
    unsigned char* img
        = stbi_load(filename, &width, &height, &channels, 3); // Load as RGB
    glm::vec3* colors = new glm::vec3[width * height];
    for (int i = 0; i < width * height; ++i) {
        colors[i] = glm::vec3(
            img[i * 3] / 255.0f,
            img[i * 3 + 1] / 255.0f,
            img[i * 3 + 2] / 255.0f
        );
    }
    CubeMap* cubeMap = new CubeMap();
    cubeMap->width = width;
    cubeMap->height = height;
    glm::vec3* data;
    hipMalloc(&data, sizeof(glm::vec3) * width * height);
    cubeMap->data = data;
    cubeMap->width = width;
    cubeMap->height = height;
    hipMemcpy(
        cubeMap->data,
        colors,
        sizeof(glm::vec3) * width * height,
        hipMemcpyHostToDevice
    );

    CubeMap* cubeMapDevice;
    hipMalloc(&cubeMapDevice, sizeof(CubeMap));
    hipMemcpy(cubeMapDevice, cubeMap, sizeof(CubeMap), hipMemcpyHostToDevice);

    free(colors);
    free(img);
    free(cubeMap);

    return cubeMapDevice;
    stbi_image_free(img);
}

inline __device__ double SpherePrimitive_SDF(
    const SpherePrimitive* sphere,
    const glm::vec4& p,
    const glm::vec4& positionWorldSpace
) {
    const glm::vec3 euclideanPosition{
        glm::vec3(positionWorldSpace) + sphere->position
    };

    const glm::vec4 hyperbolicPosition{CUDAMath::constructHyperboloidPoint(
        euclideanPosition, glm::length(euclideanPosition)
    )};

    const float dist = CUDAMath::hyperbolicSphereSDF(
        p, // todo: is w supposed to be 0?
        sphere->radius_dynamic,
        hyperbolicPosition
    );

    return dist;
}

__device__ glm::vec3 intersection_evaluate_outgoing(
    const CUDAStruct::Intersection* intersection
) {
    glm::vec3 outgoing{0.f};

    // if (reflected)
    {
        const glm::vec3 lambert{
            CUDAMath::randomHemisphereDir(0, intersection->normal)
        };
        const glm::vec3 mirror{
            glm::reflect(intersection->incidentDir, intersection->normal)
        };

        outgoing = CUDAMath::lerp(intersection->mat_roughness, mirror, lambert);
    }
    // else // light diffused or refracted
    {}

    return outgoing;
}
} // namespace CUDAStruct

namespace RendererCUDA
{

void check_device() {
    printf("Checking CUDA device...\n");
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
        printf(
            "hipGetDeviceCount returned %d\n-> %s\n",
            static_cast<int>(error_id),
            hipGetErrorString(error_id)
        );
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }

    // This function call returns 0 if there are no CUDA capable devices.
    if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA\n");
    } else {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }
}

__device__ glm::vec3 environmentalLight(
    const glm::vec3& dir,
    const CUDAStruct::Scene* scene
) {

    glm::vec3 lightDir
        = glm::vec3(sinf(scene->dayTime), cosf(scene->dayTime), 0);
    lightDir = glm::normalize(lightDir);

    const glm::vec3 noonColor{0.5};

    const glm::vec3 sunsetColor{0.5, .3, .15};

    const float interpolation
        = glm::max(0.f, glm::dot(lightDir, glm::vec3{0, 1, 0}));

    const glm::vec3 lightColor
        = CUDAMath::lerp(interpolation, sunsetColor, noonColor);

    auto light_intensity = glm::dot(lightDir, dir);

    glm::vec3 light{glm::max(0.f, glm::dot(lightDir, dir)) * lightColor};

    return light;
}

// entirely written by chatgipidy
__device__ glm::vec2 DirectionToEnvMapCoords(const glm::vec3& direction) {
    // Convert direction to spherical coordinates
    float longitude = atan2(
        direction.z, direction.x
    ); // Angle in XY plane from positive X axis
    float latitude = acos(direction.y); // Angle from positive Y axis

    // Normalize longitude to range [0, 2*pi)
    if (longitude < 0.0f)
        longitude += glm::two_pi<float>();

    // Map spherical coordinates to uv coordinates [0, 1]
    float u = longitude / glm::two_pi<float>();
    float v = 1.0f - latitude / glm::pi<float>();

    u = glm::clamp(u, 0.0f, 1.0f);
    v = glm::clamp(v, 0.0f, 1.0f);

    return glm::vec2(u, v);
}

__device__ glm::vec3 sample_environment_map(
    const glm::vec3& dir,
    const CUDAStruct::Scene* scene
) {
    glm::vec2 directionUV = DirectionToEnvMapCoords(dir);

    int index
        = (int)(directionUV.y * scene->cubemap->height) * scene->cubemap->width
          + (int)(directionUV.x * scene->cubemap->width);

    if (index >= scene->cubemap->width * scene->cubemap->height)
        return glm::vec3(0.0f, 0.0f, 0.0f);

    return scene->cubemap->data[index];
}

__device__ glm::vec3 evaluate_light_path(
    glm::vec3 origin,
    CUDAStruct::Intersection* hits,
    int num_hits,
    const CUDAStruct::Scene* scene
) {

    if (num_hits != 0) {
        // printf("Num hits: %d\n", num_hits);
    }
    glm::vec3 incomingLight{0};

    // if ray bounced off a surface and never hit anything after
    const bool reachedEnvironment{num_hits < MAX_NUM_BOUNCES};

    // if primary ray hits nothing, use that as environment bound vector
    const glm::vec3 environmentDir{
        num_hits == 0 ? origin : hits[num_hits - 1].outgoingDir
    };

    if (reachedEnvironment) {
        // if (environmentDir != glm::vec3{0}) {
        // printf("Environment dir: %f, %f, %f\n", environmentDir.x,
        // environmentDir.y, environmentDir.z);
        // }
        incomingLight += environmentalLight(environmentDir, scene);
        incomingLight += sample_environment_map(environmentDir, scene);
        // if (incomingLight != glm::vec3{0}) {
        // printf("Environment light: %f, %f, %f\n", incomingLight.x,
        //        incomingLight.y, incomingLight.z);
        // }
    }

    // reverse iterate from the start of a path of light
    for (int i = num_hits - 1; i >= 0; i--) {
        const CUDAStruct::Intersection* hit = hits + i;

        // light emitted from hit surface
        const glm::vec3 emittedLight
            = hit->mat_emissionStrength * hit->mat_emissionColor;

        // cos(theta) term
        const float lightStrength{
            1 // glm::max(0.f, glm::dot(hit.normal, -hit.incidentDir))
        };

        // basically the rendering equation
        // incomingLight = emittedLight + (2.f * Math::BRDF(hit) * incomingLight
        // * lightStrength);
        incomingLight
            = emittedLight + (hit->mat_albedo * incomingLight * lightStrength);
    }

    // printf("Incoming light: %f, %f, %f\n", incomingLight.x, incomingLight.y,
    //        incomingLight.z);
    return incomingLight;
}

// TODO: no kd tree traversal yet
__device__ void getClosestPrimitive(
    const glm::vec4& p,
    const CUDAStruct::Scene* scene,
    double* distance,
    const CUDAStruct::SpherePrimitive** closestPrimitive
) {

    for (int i = 0; i < scene->num_geometries; i++) {
        const CUDAStruct::Geometry* object = scene->geometries + i;
        const glm::vec4 objHypPos{CUDAMath::constructHyperboloidPoint(
            object->position, glm::length(object->position)
        )};
        for (int j = 0; j < object->num_spheres; j++) {
            const CUDAStruct::SpherePrimitive* sphere = object->spheres + j;
            const double d
                = CUDAStruct::SpherePrimitive_SDF(sphere, p, objHypPos);
            if (d < *distance) {
                *distance = d;
                *closestPrimitive = sphere;
                // printf("Closest primitive found at %f, %f, %f\n", p.x, p.y,
                // p.z);
            }
        }
    }
}

__device__ double getClosestDistance(
    const glm::vec4& p,
    const CUDAStruct::Scene* scene
) {
    double minDistance{1000000};
    for (int i = 0; i < scene->num_geometries; i++) {
        const CUDAStruct::Geometry* object = scene->geometries + i;
        const glm::vec4 objHypPos{CUDAMath::constructHyperboloidPoint(
            object->position, glm::length(object->position)
        )};

        for (int j = 0; j < object->num_spheres; j++) {
            const CUDAStruct::SpherePrimitive* sphere = object->spheres + j;
            const double d
                = CUDAStruct::SpherePrimitive_SDF(sphere, p, objHypPos);
            minDistance = glm::min(minDistance, d);
        }
    }
    return minDistance;
}

__device__ glm::vec3 computeNormal(
    const glm::vec4& p,
    const CUDAStruct::Scene* scene
) {

    static constexpr float EPSILON{0.001f};
    // hyperbolic normalization
    // Compute basis vectors for the tangent hyperplane at p
    glm::vec4 basis_x = CUDAMath::hypNormalize(glm::vec4(p.w, 0.0f, 0.0f, p.x));
    glm::vec4 basis_y = glm::vec4(0.0f, p.w, 0.0f, p.y);
    glm::vec4 basis_z = glm::vec4(0.0f, 0.0f, p.w, p.z);

    // Gram-Schmidt orthogonalization
    basis_y = CUDAMath::hypNormalize(
        basis_y - CUDAMath::hypDot(basis_y, basis_x) * basis_x
    );
    basis_z = CUDAMath::hypNormalize(
        basis_z - CUDAMath::hypDot(basis_z, basis_x) * basis_x
        - CUDAMath::hypDot(basis_z, basis_y) * basis_y
    );

    // Compute gradients using finite differences
    float xGradient = getClosestDistance(p + EPSILON * basis_x, scene)
                      - getClosestDistance(p - EPSILON * basis_x, scene);
    float yGradient = getClosestDistance(p + EPSILON * basis_y, scene)
                      - getClosestDistance(p - EPSILON * basis_y, scene);
    float zGradient = getClosestDistance(p + EPSILON * basis_z, scene)
                      - getClosestDistance(p - EPSILON * basis_z, scene);

    // Construct the normal vector
    glm::vec4 normal = CUDAMath::hypNormalize(
        xGradient * basis_x + yGradient * basis_y + zGradient * basis_z
    );

    return normal;
}

// Get the closest intersection, returns true if hit something and stores the
// intersection in the buffer
__device__ bool get_closest_intersection(
    glm::vec3 ray_origin,
    glm::vec3 ray_dir,
    CUDAStruct::Intersection*
        intersection_buffer, // can directly store the intersection into
    const CUDAStruct::Scene* scene,
    float hypCamPosX,
    float hypCamPosY,
    float hypCamPosZ,
    float hypCamPosW
) {
    //
    //
    float totalDistanceTraveled = 0.0;
    const int MAX_NUM_STEPS = 8;
    const float MIN_HIT_DISTANCE = .01;
    const float MAX_TRACE_DISTANCE = 20; // max float value on order of 10e38

    // translate camera position from euclidean to hyperbolic (translated to
    // hyperboloid)
    glm::vec4 hypPos{
        CUDAMath::constructHyperboloidPoint(ray_origin, glm::length(ray_origin))
    };

    const glm::vec4 p{hypCamPosX, hypCamPosY, hypCamPosZ, hypCamPosW};
    //
    // generate direction then transform to hyperboloid
    const glm::vec4 hyperbolicPos{
        p // Math::correctH3Point(p)
    };
    //   //
    const glm::vec4 d{ray_dir, 0};
    //
    const glm::vec4 hyperbolicDir{CUDAMath::correctDirection(p, d)};
    //
    glm::vec4 marchPos{hyperbolicPos};
    glm::vec4 marchDir{hyperbolicDir};
    //   //
    for (int i = 0; i < MAX_NUM_STEPS; ++i) {
        if (!CUDAMath::isH3Point(marchPos)
            || !CUDAMath::isH3Dir(marchPos, marchDir)) {
            // hyperbolicErrorAcc++; TODO: implement error handling
        }
        //       //
        //       //
        double dist = DBL_MAX;
        const CUDAStruct::SpherePrimitive* closestPrimitive = nullptr;

        getClosestPrimitive(marchPos, scene, &dist, &closestPrimitive);

        // if (closestPrimitive == nullptr) {
        //     printf("Boutta crash\n");
        // }
        //
        //       double dist = closest.first;
        //       // we hit something

        if (dist < MIN_HIT_DISTANCE) {
            // glm::vec3 normal
            //     = primitive.material.get()
            //           ->albedo; // for rough quick rendering/debugging

            // printf("Hit something at %f, %f, %f\n", marchPos.x,
            // marchPos.y, marchPos.z);
            glm::vec3 normal{0}; // TODO: implemenet normal computation

            // if (!RENDER_WITH_POTATO_SETTINGS)
            normal = computeNormal(marchPos, scene);

            // populate intersection buffer

            CUDAStruct::Intersection* intersection = intersection_buffer;
            intersection->position = marchPos;
            intersection->normal = normal;
            intersection->incidentDir = ray_dir;
            intersection->outgoingDir
                = CUDAStruct::intersection_evaluate_outgoing(intersection);

            intersection->mat_albedo = closestPrimitive->mat_albedo;
            intersection->mat_emissionColor
                = closestPrimitive->mat_emissionColor;
            intersection->mat_emissionStrength
                = closestPrimitive->mat_emissionStrength;
            intersection->mat_roughness = closestPrimitive->mat_roughness;

            return true;
        } else if (!CUDAMath::isH3Point(marchPos) 
                || !CUDAMath::isH3Dir(marchPos, marchDir) 
                || totalDistanceTraveled + dist > MAX_TRACE_DISTANCE 
                || glm::isnan(marchPos.x) 
                || glm::isnan(marchDir.x))
        {
            break;
        } else {
            const float ss{(float)dist / 1}; // substep size
            while (dist > 0) {
                glm::vec4 new_pos;
                glm::vec4 new_dir;

                // march the ray forward
                CUDAMath::geodesicFlowHyperbolic(
                    marchPos, marchDir, ss, &new_pos, &new_dir
                );

                marchPos = CUDAMath::correctH3Point(new_pos);
                marchDir = CUDAMath::correctDirection(marchPos, new_dir);
                totalDistanceTraveled += ss;
                dist -= ss;
            }
        }
    }

    return false;
    //
    //   if (closestHit)
    //       return true;
    //
    //   return false;
}

// trace a single ray and return the color
__device__ glm::vec3 trace_ray(
    glm::vec3 origin,
    glm::vec3 direction,
    CUDAStruct::Intersection* hitsBuffer_ray, // guaranteed to allow for storing
                                              // num_bounces intersection
    int num_bounces,
    const CUDAStruct::Scene* scene,
    float hypCamPosX,
    float hypCamPosY,
    float hypCamPosZ,
    float hypCamPosW
) {

    int num_hits = 0;
    for (int i = 0; i < num_bounces; i++) {
        CUDAStruct::Intersection* hitsBuffer_bounce
            = hitsBuffer_ray
              + i; // if intersection happens, store it into this buffer

        bool hit = get_closest_intersection(
            origin,
            direction,
            hitsBuffer_bounce,
            scene,
            hypCamPosX,
            hypCamPosY,
            hypCamPosZ,
            hypCamPosW

        );

        if (!hit) {
            break;
        }

        // update ray origin and direction
        origin = hitsBuffer_bounce->position
                 + hitsBuffer_bounce->outgoingDir * 0.02f;
        direction = hitsBuffer_bounce->outgoingDir;

        num_hits++;
    }

    CUDAStruct::Intersection* hits = hitsBuffer_ray;
    const glm::vec3 finalColor{
        evaluate_light_path(direction, hits, num_hits, scene)
    };

    return finalColor;
}

// Render a single pixel
__global__ void render_pixel(
    const CUDAStruct::Scene* scene,
    const Camera* camera,
    int width,
    int height,
    int rays_per_pixel,
    int bounces_per_ray,
    glm::vec3* frameBuffer_device,
    CUDAStruct::Intersection* hitsBuffer_device,
    float hypCamPosX,
    float hypCamPosY,
    float hypCamPosZ,
    float hypCamPosW
) {
    // printf("Rendering pixel. Width: %d, Height: %d, rays_per_pixel: %d,
    // bounces_per_ray: %d\n", width, height, rays_per_pixel, bounces_per_ray);

    // TODO: these should be passed in as parameters
    float aspectRatio = width / height; // w : h
    // todo figure out why FOV seems "off"
    float fovComponent{tanf(camera->FOV / 2.f)};

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // check if out of bounds
    if (x >= width || y >= height) {
        return;
    }

    // printf("Rendering pixel at x: %d, y: %d\n", x, y);

    CUDAStruct::Intersection* hitsBuffer_pixel
        = hitsBuffer_device
          + (x + (y * width) * rays_per_pixel * bounces_per_ray);

    glm::vec3 final_color{0.f};

    const glm::vec2 ndc{(x + 0.5f) / width, (y + 0.5f) / height};

    glm::uvec2 pixelCoord{ndc.x * width, ndc.y * height};

    for (int i = 0; i < rays_per_pixel; i++) {
        float2 rayOffset = CUDAMath::randomVec2(
            i + (x * width) + y
        ); // TODO: not sure if rng works

        const glm::vec2 ndcAliased{
            (x + rayOffset.x) / width, (y + rayOffset.y) / height
        };

        // screen space
        glm::vec2 coord = glm::vec2{
            ((2.f * ndc.x) - 1.f) * fovComponent * aspectRatio,
            1.f - (2.f * ndc.y) * fovComponent // flip vertically so +y is up
        };

        // ray coords in world space
        glm::vec4 start{camera->position, 1.f};
        glm::vec4 dir{coord.x, coord.y, -1.f, 0};

        // transform ray to view space
        dir = glm::normalize(dir);
        dir = dir * camera->viewMat;

        CUDAStruct::Intersection* hitsBuffer_ray
            = hitsBuffer_pixel + (i * bounces_per_ray);

        glm::vec3 color = trace_ray(
            start,
            dir,
            hitsBuffer_ray,
            bounces_per_ray,
            scene,
            hypCamPosX,
            hypCamPosY,
            hypCamPosZ,
            hypCamPosW
        );

        final_color += color;
    }

    final_color /= RAYS_PER_PIXEL;

    { // writeback to framebuffer
        int frameBufferIndex = x + (y * width);
        frameBuffer_device[frameBufferIndex] = final_color;
    }
}

__host__ void render(
    const CUDAStruct::Scene* scene,
    const Camera* camera,
    Image* image
) {
    // printf("Rendering with CUDA\n");
    print_cuda_error();
    float aspectRatio = (float)image->width / image->height; // w : h

    // todo figure out why FOV seems "off"
    float fovComponent{tanf(camera->FOV / 2.f)};

    int width = image->width;
    int height = image->height;

    dim3 blockDims = dim3(16, 16); // 256 threads per block
    dim3 gridDims = dim3(
        (width + blockDims.x - 1) / blockDims.x,
        (height + blockDims.y - 1) / blockDims.y
    );

    // allocate FB
    glm::vec3* frameBuffer = image->pixels.data();
    glm::vec3* frameBuffer_Device;
    size_t frameBuffer_size = width * height;
    hipMalloc(&frameBuffer_Device, frameBuffer_size * sizeof(glm::vec3));

    // allocate buffer to store intersections data
    CUDAStruct::Intersection* hitsBuffer_Device;
    hipMalloc(
        &hitsBuffer_Device,
        width * height * RAYS_PER_PIXEL * MAX_NUM_BOUNCES
            * sizeof(CUDAStruct::Intersection)
    ); // each ray (bounce) needs to store its hit

    // allocate mem for cudascene
    CUDAStruct::Scene* scene_Device;
    hipMalloc(&scene_Device, sizeof(CUDAStruct::Scene));
    hipMemcpy(
        scene_Device, scene, sizeof(CUDAStruct::Scene), hipMemcpyHostToDevice
    );

    // allocate camera
    Camera* camera_Device;
    hipMalloc(&camera_Device, sizeof(Camera));
    hipMemcpy(camera_Device, camera, sizeof(Camera), hipMemcpyHostToDevice);

    render_pixel<<<gridDims, blockDims>>>(
        scene_Device,
        camera_Device,
        width,
        height,
        RAYS_PER_PIXEL,
        MAX_NUM_BOUNCES,
        frameBuffer_Device,
        hitsBuffer_Device,
        // FIXME: these values shouldn't be passed as params, instead store them
        // in camera
        hypCamPosX,
        hypCamPosY,
        hypCamPosZ,
        hypCamPosW
    );

    hipDeviceSynchronize();

    hipMemcpy(
        frameBuffer,
        frameBuffer_Device,
        width * height * sizeof(glm::vec3),
        hipMemcpyDeviceToHost
    );
    hipFree(frameBuffer_Device);
    hipFree(hitsBuffer_Device);
    hipFree(scene_Device);
    hipFree(camera_Device);

    // print_cuda_error();
    // printf("Finished rendering with CUDA\n");
}

} // namespace RendererCUDA

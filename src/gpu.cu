#include "hip/hip_runtime.h"
#include "gpu.h"
#include <iostream>
#include <stdio.h>
#include <glm/glm.hpp>


#include "Camera.h"
#include "Image.h"
#include "Scene.h"

namespace CudaPlayground {
__global__ void cudaHello() {
    printf("Hello World from CUDA thread [%d,%d]\n", threadIdx.x, blockIdx.x);
    glm::vec3 a(1.0f, 2.0f, 3.0f);
    glm::vec3 b(4.0f, threadIdx.x, blockIdx.x);
    glm::vec3 c = a + b;
    printf("c = [%f, %f, %f]\n", c.x, c.y, c.z);
}

void play() {
    std::cout << "Running CUDA Playground" << std::endl;
    int numBlocks = 16;
    int threadsPerBlock = 16;
    cudaHello<<<numBlocks, threadsPerBlock>>>();
    hipDeviceSynchronize();

    std::cout << "CUDA Playground finished" << std::endl;
}
} // namespace CudaPlayground

namespace RendererCUDA {

__global__ void _render(const Scene* scene, const Camera* camera, Image* image) {
    float aspectRatio = (float)image->width / image->height; // w : h

    // todo figure out why FOV seems "off"
    float fovComponent{tanf(camera->FOV / 2.f)};

    // for (int y = 0; y < image.height; ++y) {
    //     for (int x = 0; x < image.width; ++x) {
    //         const int index = x + (y * image.width);
    //
    //         // misc debug stuff
    //         const glm::uvec2 debugRay{image.width / 2, image.height / 2};
    //         isDebugRay = (x == debugRay.x && y == debugRay.y);
    //
    //         if constexpr (!INTERACTIVE_MODE) {
    //             const unsigned index{x + (y * image.width)};
    //             const unsigned numPixels{image.width * image.height};
    //
    //             const float completionPercent{100.f * index / numPixels};
    //
    //             // how many pixels per print
    //             constexpr unsigned printFreq{50};
    //
    //             static int prevPrintIndex{0};
    //
    //             if (index > prevPrintIndex + printFreq) {
    //                 prevPrintIndex = index;
    //                 std::cout << completionPercent << "%\n";
    //             }
    //         }
    //
    //         // ray tracing stuff
    //         const glm::vec2 ndc{(x + 0.5f) / image.width, (y + 0.5f) / image.height};
    //
    //         glm::vec3 color{0.f};
    //
    //         for (int i = 0; i < RAYS_PER_PIXEL; ++i) {
    //             glm::vec2 rayOffset = Math::randomVec2(rngSeed + i);
    //
    //             const glm::vec2 ndcAliased{(x + rayOffset.x) / image.width, (y + rayOffset.y) / image.height};
    //
    //             // screen space
    //             glm::vec2 coord;
    //
    //             if constexpr (ANTIALIAS) {
    //                 coord = glm::vec2{
    //                     ((2.f * ndcAliased.x) - 1.f) * fovComponent * aspectRatio,
    //                     1.f - (2.f * ndcAliased.y) * fovComponent // flip vertically so +y is up
    //                 };
    //             } else {
    //                 coord = glm::vec2{
    //                     ((2.f * ndc.x) - 1.f) * fovComponent * aspectRatio,
    //                     1.f - (2.f * ndc.y) * fovComponent // flip vertically so +y is up
    //                 };
    //             }
    //
    //             // ray coords in world space
    //             glm::vec4 start{camera.position, 1.f};
    //             glm::vec4 dir{coord.x, coord.y, -1.f, 0};
    //
    //             // transform ray to view space
    //             dir = glm::normalize(dir);
    //             dir = dir * camera.viewMat;
    //
    //             Ray ray{start, dir};
    //
    //             if (!accumulate)
    //                 resetAccumulator();
    //
    //             // isDebugRay = index == 3846;
    //
    //             color += traceRay(ray, scene);
    //
    //             frameBuffer[index] += color;
    //         }
    //
    //         glm::vec3 pixelColor{frameBuffer[index]};
    //
    //         // average color
    //         pixelColor /= RAYS_PER_PIXEL;
    //
    //         // normalize color
    //         pixelColor.r = std::clamp(pixelColor.r, 0.f, 1.f);
    //         pixelColor.g = std::clamp(pixelColor.g, 0.f, 1.f);
    //         pixelColor.b = std::clamp(pixelColor.b, 0.f, 1.f);
    //
    //         // debug visualization
    //         const bool shouldInvertColor{
    //             VISUALIZE_DEBUG_RAY && ((x == debugRay.x + 1 && y == debugRay.y + 0) || // left
    //                                     (x == debugRay.x - 1 && y == debugRay.y - 0) || // right
    //                                     (x == debugRay.x + 0 && y == debugRay.y + 1) || // top
    //                                     (x == debugRay.x - 0 && y == debugRay.y - 1))   // bottom
    //         };
    //
    //         if (shouldInvertColor)
    //             pixelColor = glm::vec3{1} - pixelColor;
    //
    //         // actually setting pixel
    //         image.setPixel(ndc, pixelColor);
    //     }
    // }

}

} // namespace RendererCUDA
